// This program is for accelerating the manhattan distance calculation
// between pairs of basic block vectors
// Only works for single kernel files (#TODO later)
// Maybe try GNU plot? (#TODO but Python is good for pretty pictures)
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <string>
#include <sstream>
#include <iostream>
#include <fstream>
#include <assert.h>

using namespace std;

// GPU kernel for calculating the Manhattan distance between basic block
// vectors. (#TODO easy candidate for cache tiling)
// Takes:
//  basic_blocks:   Pointer to basic block vectors
//  distances:      Pointer to Manhattan distance results
//  n_bbs    :      Number of basic blocks per basic block vector
//  n_threads:      Total number of basic block vectors
__global__ void m_distance(unsigned *basic_blocks, unsigned *distances, unsigned n_bbs, unsigned n_threads){
    // Calculate global thread position
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if(tid < n_threads){
        // Temp variable for each distance calculation
        unsigned temp;

        // Compare this thread's BBV with all others
        for(int i = 0; i < n_threads; i++){
            // Reset temp between BBVs
            temp = 0;
            // Find distance between the two basic block counts
            for(int j = 0; j < n_bbs; j++){
                // Use sum of absolute difference intrinsic
                temp = __usad(basic_blocks[tid * n_bbs + j],
                        basic_blocks[i * n_bbs + j], temp);
            }
            
            // Write back the distance
            distances[tid * n_threads + i] = temp;
        }
    }
}

// Function for reading basic blocks from files into the program
// Takes:
//  data_file:      file pointer passed by reference
//  n:              number of integers to read
//  basic_blocks:   array storing the read in basic basic block counts
void read_file(ifstream &data_file, unsigned n_bbs, unsigned n_threads,
        unsigned *basic_blocks){
    for(int i = 0; i < (n_bbs * n_threads); i++){
        data_file >> basic_blocks[i];
    }
}

// Function for writing the Manhattan distances to a new file
// Takes:
//  output_file:    file pointer passed by reference
//  n_threads:      number of threads that had distances compared
//  distances:      array storing the Manhattan distances
void write_file(ofstream &output_file, unsigned n_threads, unsigned *distances){
    for(int i = 0; i < (n_threads * n_threads); i++){
        output_file << distances[i] << " ";
    }
    output_file << endl;
}

int main(int argc, char *argv[]){
    // Check if a file was passed in as an argument
    if(argc != 2){
        cout << "ERROR: No data file passed as an argument" << endl;
        assert(false);
    }

    // Open the file passed in as an argument
    ifstream data_file;
    data_file.open(argv[1]);

    // Check if the file was opened successfully
    if(!data_file){
        cout << "ERROR: Can not open file with path: " << argv[1]
            << endl;
        assert(false);
    }

    // Variables to read in for each kernel of an app
    string kernel_name;
    unsigned n_bbs = 0;
    unsigned n_warps = 0;

    // Unified memory pointer
    unsigned *basic_blocks;

    // Unique kernel id
    int kid = 0;
    // While we can still read in a kernel name
    while(getline(data_file, kernel_name)){
        // Dumb error check to break on empty kernel name (EOF?)
        if(kernel_name == "")
            continue;

        // We should then get #BBs and #warps
        data_file >> n_warps;
        data_file >> n_bbs;

        // Allocate space for all the basic blocks
        hipMallocManaged(&basic_blocks, n_bbs * n_warps * sizeof(unsigned));

        // Read out the basic block distributions
        read_file(data_file, n_bbs, n_warps, basic_blocks);

        // Allocate space for the basic block differences
        unsigned *distances;
        hipMallocManaged(&distances, n_warps * n_warps * sizeof(unsigned));

        // Calculate grid dimensions using 512 thread TBs
        int TB_SIZE = 512;
        int GRID_SIZE = (n_warps + TB_SIZE - 1) / TB_SIZE;

        // Launch the kernel
        m_distance<<<GRID_SIZE, TB_SIZE>>>(basic_blocks, distances, n_bbs, n_warps);
    
        // Wait for the kernel to complete
        hipDeviceSynchronize();

        // Open a file based on the kernel's name (truncate if exists)
        string output_name = kernel_name.substr(0, kernel_name.find("("));
        stringstream ss;
        ss << kid;
        output_name.append("_" + ss.str() + ".txt");
        ofstream output_file;
        output_file.open(output_name.c_str(), ios::out | ios::app);
    
        // Add header to the output file
        output_file << kernel_name << endl;
        output_file << n_bbs << endl;
        output_file << n_warps << endl;

        // Write the output to a similarly formatted separate file
        write_file(output_file, n_warps, distances);

        // Close the output file
        output_file.close();

        // De-allocate unified memory
        hipFree(basic_blocks);

        // Increase kernel ID
        kid++;
    }

    // Close the data file
    data_file.close();

    return 0;
}

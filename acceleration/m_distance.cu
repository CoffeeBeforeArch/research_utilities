// This program is for accelerating the manhattan distance calculation
// between pairs of basic block vectors
// Only works for single kernel files (#TODO later)
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <fstream>
#include <assert.h>

using namespace std;

__global__ void m_distance(){

}

// Function for reading basic blocks from files into the program
// Takes:
//  data_file:      file pointer passed by reference
//  n:              number of integers to read
//  basic_blocks:   array storing the read in basic basic block counts
void read_file(ifstream &data_file, int n_bbs, int n_threads, int *basic_blocks){
    for(int i = 0; i < (n_bbs * n_threads); i++){
        data_file >> basic_blocks[i];
    }
}

int main(int argc, char *argv[]){
    // Check if a file was passed in as an argument
    if(argc != 2){
        cout << "ERROR: No data file passed as an argument" << endl;
        assert(false);
    }

    // Open the file passed in as an argument
    ifstream data_file;
    data_file.open(argv[1]);

    // Check if the file was opened successfully
    if(!data_file){
        cout << "ERROR: Can not open file with path: " << argv[1]
            << endl;
        assert(false);
    }

    // Read kernel name, # basic blocks, and # threads
    string kernel_name;
    int n_bbs = 0;
    int n_threads = 0;
    data_file >> kernel_name;
    data_file >> n_bbs;
    data_file >> n_threads;

    // Allocate space for all the basic blocks
    int *basic_blocks  = new int[n_bbs * n_threads];

    // Read out the basic block distributions
    read_file(data_file, n_bbs, n_threads, basic_blocks);

    return 0;
}

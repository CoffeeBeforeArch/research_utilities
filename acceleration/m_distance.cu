// This program is for accelerating the manhattan distance calculation
// between pairs of basic block vectors
// Only works for single kernel files (#TODO later)
// Maybe try GNU plot? (#TODO but Python is good for pretty pictures)
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <fstream>
#include <assert.h>

using namespace std;

// GPU kernel for calculating the Manhattan distance between basic block
// vectors. (#TODO easy candidate for cache tiling)
// Takes:
//  basic_blocks:   Pointer to basic block vectors
//  distances:      Pointer to Manhattan distance results
//  n_bbs    :      Number of basic blocks per basic block vector
//  n_threads:      Total number of basic block vectors
__global__ void m_distance(unsigned *basic_blocks, unsigned *distances, unsigned n_bbs, unsigned n_threads){
    // Calculate global thread position
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if(tid < n_threads){
        // Temp variable for each distance calculation
        unsigned temp;

        // Compare this thread's BBV with all others
        for(int i = 0; i < n_threads; i++){
            // Reset temp between BBVs
            temp = 0;
            // Find distance between the two basic block counts
            for(int j = 0; j < n_bbs; j++){
                // Use sum of absolute difference intrinsic
                temp = __usad(basic_blocks[tid * n_bbs + j],
                        basic_blocks[i * n_bbs + j], temp);
            }
            
            // Write back the distance
            distances[tid * n_threads + i] = temp;
        }
    }
}

// Function for reading basic blocks from files into the program
// Takes:
//  data_file:      file pointer passed by reference
//  n:              number of integers to read
//  basic_blocks:   array storing the read in basic basic block counts
void read_file(ifstream &data_file, unsigned n_bbs, unsigned n_threads,
        unsigned *basic_blocks){
    for(int i = 0; i < (n_bbs * n_threads); i++){
        data_file >> basic_blocks[i];
    }
}

// Function for writing the Manhattan distances to a new file
// Takes:
//  output_file:    file pointer passed by reference
//  n_threads:      number of threads that had distances compared
//  distances:      array storing the Manhattan distances
void write_file(ofstream &output_file, unsigned n_threads, unsigned *distances){
    for(int i = 0; i < (n_threads * n_threads); i++){
        output_file << distances[i] << " ";
    }
    output_file << endl;
}

int main(int argc, char *argv[]){
    // Check if a file was passed in as an argument
    if(argc != 2){
        cout << "ERROR: No data file passed as an argument" << endl;
        assert(false);
    }

    // Open the file passed in as an argument
    ifstream data_file;
    data_file.open(argv[1]);

    // Check if the file was opened successfully
    if(!data_file){
        cout << "ERROR: Can not open file with path: " << argv[1]
            << endl;
        assert(false);
    }

    // Read kernel name, # basic blocks, and # threads
    string kernel_name;
    unsigned n_bbs = 0;
    unsigned n_threads = 0;
    data_file >> kernel_name;
    data_file >> n_bbs;
    data_file >> n_threads;

    // Allocate space for all the basic blocks
    unsigned *basic_blocks;
    hipMallocManaged(&basic_blocks, n_bbs * n_threads * sizeof(unsigned));

    // Read out the basic block distributions
    read_file(data_file, n_bbs, n_threads, basic_blocks);

    // Close the data file
    data_file.close();

    // Allocate space for the basic block differences
    unsigned *distances;
    hipMallocManaged(&distances, n_threads * n_threads * sizeof(unsigned));

    // Calculate grid dimensions using 512 thread TBs
    int TB_SIZE = 512;
    int GRID_SIZE = (n_threads + TB_SIZE - 1) / TB_SIZE;

    // Launch the kernel
    m_distance<<<GRID_SIZE, TB_SIZE>>>(basic_blocks, distances, n_bbs, n_threads);
    
    // Wait for the kernel to complete
    hipDeviceSynchronize();

    // Open a file based on the kernel's name (truncate if exists)
    string output_name = kernel_name;
    output_name.append("1.txt");
    ofstream output_file;
    output_file.open(output_name.c_str(), ios::out | ios::trunc);
    
    // Add header to the output file
    output_file << kernel_name << endl;
    output_file << n_bbs << endl;
    output_file << n_threads << endl;

    // Write the output to a similarly formatted separate file
    write_file(output_file, n_threads, distances);

    // Close the output file
    output_file.close();

    return 0;
}
